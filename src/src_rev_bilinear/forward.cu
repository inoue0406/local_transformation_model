#include "hip/hip_runtime.h"
#include <torch/script.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

//// threadIdx.x: batch_size
//// blockIdx.x: num_points
//// blockIdx.y: 4

__global__ void cal_pc_grid_index_kernel(
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> pc,   //// (batch, 2, num_points)  assume 0<=pc<=1
    const int grid_size,
    torch::PackedTensorAccessor<int32_t,3,torch::RestrictPtrTraits,size_t> pc_grid_index   //// (batch, 4, num_points)
    )
{
  float dx=1.0/(grid_size-1);
  float x = pc[threadIdx.x][0][blockIdx.x];
  float y = pc[threadIdx.x][1][blockIdx.x];
  int cell_x0 = __float2int_rd(x/dx), cell_x1 = cell_x0+1;
  int cell_y0 = __float2int_rd(y/dx), cell_y1 = cell_y0+1;
  cell_x1 = cell_x1 > (grid_size-1) ? (grid_size-1) : cell_x1;
  cell_y1 = cell_y1 > (grid_size-1) ? (grid_size-1) : cell_y1;
  cell_x0 = max(cell_x0, 0);
  cell_y0 = max(cell_y0, 0);
  cell_x1 = min(cell_x1, grid_size-1);
  cell_y1 = min(cell_y1, grid_size-1);
  pc_grid_index[threadIdx.x][0][blockIdx.x] = cell_x0;
  pc_grid_index[threadIdx.x][1][blockIdx.x] = cell_x1;
  pc_grid_index[threadIdx.x][2][blockIdx.x] = cell_y0;
  pc_grid_index[threadIdx.x][3][blockIdx.x] = cell_y1;
}

torch::Tensor cal_pc_grid_index(torch::Tensor pc, int grid_size)
{
  int batch_size = pc.size(0);
  int num_points = pc.size(2);
  torch::Tensor pc_grid_index = torch::zeros({batch_size, 4, num_points}).to(pc).to(at::kInt);
  ///pc = (pc + 1) / 2;
  
  const int threads = batch_size;
  const dim3 blocks(num_points, 1);

  cal_pc_grid_index_kernel<<<blocks, threads>>>(
        pc.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
        grid_size,
        pc_grid_index.packed_accessor<int32_t,3,torch::RestrictPtrTraits,size_t>()
        );

  return pc_grid_index;

}

//// blockIdx.x: num_points
//// threadIdx.x: batch_size

__global__ void cal_weight_sum_kernel(
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> pc,   //// (batch, 2, num_points)  assume 0<=pc<=1
    const torch::PackedTensorAccessor<int32_t,3,torch::RestrictPtrTraits,size_t> pc_grid_index,   //// (batch, 4, num_points)
    const int grid_size,
    torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> weight_sum   //// (batch, grid_size, grid_size)
    )
{
  float dx=1.0/(grid_size-1);
  float x = pc[threadIdx.x][0][blockIdx.x];
  float y = pc[threadIdx.x][1][blockIdx.x];
  int cell_x0 = pc_grid_index[threadIdx.x][0][blockIdx.x], cell_x1 = pc_grid_index[threadIdx.x][1][blockIdx.x];
  int cell_y0 = pc_grid_index[threadIdx.x][2][blockIdx.x], cell_y1 = pc_grid_index[threadIdx.x][3][blockIdx.x];
  float x0=cell_x0*dx, x1=cell_x1*dx, y0=cell_y0*dx, y1=cell_y1*dx;
  if(x0<x && x1>x && y0<y && y1>y){
  float w00=(x1-x) * (y1-y);
  float w01=(x1-x) * (y-y0);
  float w10=(x-x0) * (y1-y);
  float w11=(x-x0) * (y-y0);

  atomicAdd(&(weight_sum[threadIdx.x][cell_x0][cell_y0]), w00);
  atomicAdd(&(weight_sum[threadIdx.x][cell_x0][cell_y1]), w01);
  atomicAdd(&(weight_sum[threadIdx.x][cell_x1][cell_y0]), w10);
  atomicAdd(&(weight_sum[threadIdx.x][cell_x1][cell_y1]), w11);
  }
}

torch::Tensor cal_weight_sum(torch::Tensor pc, torch::Tensor pc_grid_index, int grid_size)
{
  int batch_size = pc.size(0);
  int num_points = pc.size(2);
  torch::Tensor weight_sum = torch::zeros({batch_size, grid_size, grid_size}, torch::dtype(torch::kFloat32)).cuda();
  ///pc = (pc + 1) / 2;

  const int threads = batch_size;
  const dim3 blocks(num_points, 1);

  cal_weight_sum_kernel<<<blocks, threads>>>(
        pc.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
        pc_grid_index.packed_accessor<int32_t,3,torch::RestrictPtrTraits,size_t>(),
        grid_size,
        weight_sum.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>()
        );

  return weight_sum;
}

//// blockIdx.x: num_points
//// blockIdx.y: num_channel
//// threadIdx.x: batch_size

__global__ void cal_grid_value_kernel(
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> pc,   //// (batch, 2, num_points)  assume 0<=pc<=1
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> pc_value, //// (batch, channel, num_points)
    const torch::PackedTensorAccessor<int32_t,3,torch::RestrictPtrTraits,size_t> pc_grid_index,   //// (batch, 4, num_points)
    const int grid_size,
    torch::PackedTensorAccessor<float,4,torch::RestrictPtrTraits,size_t> grid_value   //// (batch, channel, grid_size, grid_size)
    )
{
  float dx=1.0/(grid_size-1);
  float x = pc[threadIdx.x][0][blockIdx.x];
  float y = pc[threadIdx.x][1][blockIdx.x];
  int cell_x0 = pc_grid_index[threadIdx.x][0][blockIdx.x], cell_x1 = pc_grid_index[threadIdx.x][1][blockIdx.x];
  int cell_y0 = pc_grid_index[threadIdx.x][2][blockIdx.x], cell_y1 = pc_grid_index[threadIdx.x][3][blockIdx.x];
  float x0=cell_x0*dx, x1=cell_x1*dx, y0=cell_y0*dx, y1=cell_y1*dx;
  if(x0<x && x1>x && y0<y && y1>y){
    float w00=(x1-x) * (y1-y);
    float w01=(x1-x) * (y-y0);
    float w10=(x-x0) * (y1-y);
    float w11=(x-x0) * (y-y0);
    
    float value = pc_value[threadIdx.x][blockIdx.y][blockIdx.x];
    atomicAdd(&(grid_value[threadIdx.x][blockIdx.y][cell_x0][cell_y0]), w00*value);
    atomicAdd(&(grid_value[threadIdx.x][blockIdx.y][cell_x0][cell_y1]), w01*value);
    atomicAdd(&(grid_value[threadIdx.x][blockIdx.y][cell_x1][cell_y0]), w10*value);
    atomicAdd(&(grid_value[threadIdx.x][blockIdx.y][cell_x1][cell_y1]), w11*value);
    // debug print
    //printf("-------------------------\n");
    //printf("x0,x1,x:%f,%f,%f\n",x0,x1,x);
    //printf("cell_x0,cell_x1:%d,%d\n",cell_x0,cell_x1);
    //printf("y0,y1,y:%f,%f,%f\n",y0,y1,y);
    //printf("cell_y0,cell_y1:%d,%d\n",cell_y0,cell_y1);
    //printf("w00,w01,w10,w11:%f,%f,%f,%f\n",w00,w01,w10,w11);
    //printf("value:%f\n",value);
  }
}


torch::Tensor cal_grid_value(torch::Tensor pc, torch::Tensor pc_value, torch::Tensor pc_grid_index, int grid_size)
{
  int batch_size = pc.size(0);
  int num_points = pc.size(2);
  int num_channel = pc_value.size(1);
  torch::Tensor grid_value = torch::zeros({batch_size, num_channel, grid_size, grid_size}).to(pc);
  ///pc = (pc + 1) / 2;

  const int threads = batch_size;
  const dim3 blocks(num_points, num_channel);

  cal_grid_value_kernel<<<blocks, threads>>>(
        pc.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
        pc_value.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
        pc_grid_index.packed_accessor<int32_t,3,torch::RestrictPtrTraits,size_t>(),
        grid_size,
        grid_value.packed_accessor<float,4,torch::RestrictPtrTraits,size_t>()
        );

  return grid_value;
}
